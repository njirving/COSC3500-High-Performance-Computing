
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <iterator>
#include <fstream>
#include <chrono>
#include <string>
#include <iostream>
#include <stdio.h>
#include <cstring>

#define MAX_BUFFER_BYTES 400000000

using namespace std;

/**
*   Prints the current state, used for debugging
*
*/
void print_state(int* state, int rowSize) 
{
    for (auto i = 0; i < rowSize*rowSize; i++) {
        if (i % rowSize == 0) {
            std::cerr << "\n";
        }
        if (state[i] == 0) {
            std::cerr << ".";
        } else {
            std::cerr << "x";
        }
    }
    std::cerr << "\n";
}

/**
*   Initial state is read from a file and put into the state array
*
*/

void init_from_file(std::string name, int* state, int rowSize) 
{
    std::ifstream file;
    file.open(name);
    int i, j = 0;
    std::string line;
    std::getline (file, line);
    while (!file.eof()) {
        for (char c : line) {
            state[i + j * rowSize] = c - '0';
            i++;
        }
        j++;
        i = 0;
        std::getline (file, line);
    }
}

/**
*   Streams binary data to a file
*
*/
void stream_to_file(int rowSize, std::string filename, int* state, int bufftimes)
{
    std::ofstream outFile;
    int size = rowSize*rowSize*bufftimes;
    outFile.open(filename.c_str(), ios::binary | ios::app);

    for (int i = 0; i < size; i++) {
        outFile.write((char*) (&state[i]), 1);
    }

    outFile.flush();
    outFile.close();
}

/**
*   Adds state to state buffer
*
*/
void add_to_buffer(int* state, int* buffer, int rowSize, int bufftimes) 
{
    int size = rowSize*rowSize*bufftimes;
    std::memcpy(&buffer[size], state, rowSize*rowSize*sizeof(int));
}

/**
*   Prints the CUDA error to the stderr stream, shamlessly stolen from the tutorials
*
*/
void checkError(hipError_t e)
{
   if (e != hipSuccess) {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

/**
*   num_numbers transferred from the serial implementation to a device function that can
*   be run on the GPU.
*
*/
__device__ int num_neighbours(int x, int y, int rowSize, int* state) 
{
    int num = 0;
    
    int xm1 = x - 1;
    int ym1 = y - 1;
    int xp1 = x + 1;
    int yp1 = y + 1;

    if (x == 0) {
        xm1 = rowSize - 1;
    }
    if (x == rowSize - 1) {
        xp1 = 0;
    }
    if (y == 0) {
        ym1 = rowSize - 1;
    }
    if (y == rowSize - 1) {
        yp1 = 0;
    }

    num += state[xm1 + y*rowSize] 
    + state[xp1 + y*rowSize] 
    + state[x + ym1*rowSize]
    + state[x + yp1*rowSize]
    + state[xm1 + ym1*rowSize]
    + state[xp1 + yp1*rowSize]
    + state[xm1 + yp1*rowSize]
    + state[xp1 + ym1*rowSize];

    return num;
}

/**
*   CUDA kernal, handles each cell each tick
*
*/
__global__ void tick(int* state, int* savedState, int rowSize) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int x = i % rowSize;
    int y = i / rowSize;
    int num = num_neighbours(x, y, rowSize, state);
    int cell = state[x + y * rowSize];

    savedState[i] = ((num == 3 || num == 2) && cell == 1) || (num == 3 && cell == 0);
}


int main(int argc, char** argv)
{
    if (argc < 3) {
        std::cerr << "Usage: ./cudaCellular {size} {ticks} {file name}" << std::endl;
        exit(0);
    }

    //Init variables
    int rowSize = std::stoi(argv[1]);
    int ticks = std::stoi(argv[2]);
    int* deviceState;
    int* deviceSavedState;
    int* state = (int*)calloc(rowSize * rowSize, sizeof(int)); 
    int* savedState = (int*)calloc(rowSize * rowSize, sizeof(int)); 
    std::string fileName;

    int maxCopies = MAX_BUFFER_BYTES/(rowSize* rowSize);

    cerr << maxCopies << "\n";

    int* buffer = (int*)calloc(rowSize*rowSize*(maxCopies+1), sizeof(int));
    
    if (argc == 4) {
        init_from_file(argv[3], state, rowSize);
        fileName = std::string(argv[3]) + "_data_R=" + std::to_string(rowSize) + "_T=" + std::to_string(ticks) + ".bin";
    } else {
        std::srand(4);
        for (int i = 0; i < rowSize * rowSize; i++) {
            state[i] = std::rand() % 2;
        }
        fileName = "data_R=" + std::to_string(rowSize) + "_T=" + std::to_string(ticks) + ".bin";
    }

    //Removing any files with the same name as the .bin file so we don't append to random data
    remove(fileName.c_str());

    //Allocate memory on the device
    checkError(hipMalloc(&deviceState, rowSize*rowSize*sizeof(double)));
    checkError(hipMalloc(&deviceSavedState, rowSize*rowSize*sizeof(double)));

    //Copy states over to the device
    checkError(hipMemcpy(deviceState, state, rowSize*rowSize*sizeof(int), hipMemcpyHostToDevice));
    checkError(hipMemcpy(deviceSavedState, savedState, rowSize*rowSize*sizeof(int), hipMemcpyHostToDevice));

    int Threads = 256;
    int Blocks = (rowSize*rowSize + Threads - 1)/Threads;

    //Init clock
    auto start = std::chrono::high_resolution_clock::now();

    int bufftimes = 1;

    /*
    *   Main loop
    */
    for (int i = 0; i < ticks; i++) {

        add_to_buffer(state, buffer, rowSize, bufftimes);
        bufftimes++;

        if (i != 0 && (i + 1) % maxCopies == 0) {
            stream_to_file(rowSize, fileName, buffer, bufftimes);
            bufftimes = 0;
        }

        tick<<<Blocks, Threads>>>(deviceState, deviceSavedState, rowSize);
        checkError(hipDeviceSynchronize());

        checkError(hipMemcpy(state, deviceState, rowSize*rowSize*sizeof(int), hipMemcpyDeviceToHost));
        checkError(hipMemcpy(savedState, deviceSavedState, rowSize*rowSize*sizeof(int), hipMemcpyDeviceToHost));

        std::swap(state, savedState);

        checkError(hipMemcpy(deviceState, state, rowSize*rowSize*sizeof(int), hipMemcpyHostToDevice));
        checkError(hipMemcpy(deviceSavedState, savedState, rowSize*rowSize*sizeof(int), hipMemcpyHostToDevice));
    }

    checkError(hipMemcpy(state, deviceState, rowSize*rowSize*sizeof(int), hipMemcpyDeviceToHost));
    checkError(hipMemcpy(savedState, deviceSavedState, rowSize*rowSize*sizeof(int), hipMemcpyDeviceToHost));
    
    stream_to_file(rowSize, fileName, buffer, bufftimes);

    //Stop clock and get diff, print diff to the stdout stream
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "R=" << argv[1] << " T=" << argv[2] << " " << duration << "ms" << std::endl;

    //Free Mem
    hipFree(deviceState);
    hipFree(deviceSavedState);
    free(savedState);
    free(state);
}
